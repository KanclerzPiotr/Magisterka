#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>


#pragma nv_diagnostic push
#pragma nv_diag_suppress 550
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#pragma nv_diag_default 550
#pragma nv_diagnostic pop

constexpr int kernel_size = 5;
constexpr int padding     = kernel_size / 2;
constexpr int coeff       = 273;
constexpr int kernel[kernel_size * kernel_size] = 
      {1,  4,  7,  4, 1,
       4, 16, 26, 16, 4,
       7, 26, 41, 26, 7,
       4, 16, 26, 16, 4,
       1,  4,  7,  4, 1};

__constant__ int d_kernel[kernel_size * kernel_size];

// CUDA kernel: 5x5 Gaussian blur on uchar4 image with padding
__global__ void gaussianBlur(
    const uchar4* __restrict__ paddedData,
          uchar4* __restrict__ targetData,
    int width,
    int height,
    int pitchElems        // <-- elements per row
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (ix >= width || iy >= height) return;

    int x     = ix + padding;
    int y     = iy + padding;
    int coord = y*pitchElems + x;    // now correct

    uint4 sum = make_uint4(0,0,0,0);
    for(int ky=-padding; ky<=padding; ++ky){
      for(int kx=-padding; kx<=padding; ++kx){
        int ki  = (ky+padding)*kernel_size + (kx+padding);
        int off = ky*pitchElems + kx;
        uchar4 pix = paddedData[coord + off];
        sum.x += d_kernel[ki]*pix.x;
        sum.y += d_kernel[ki]*pix.y;
        sum.z += d_kernel[ki]*pix.z;
        sum.w += d_kernel[ki]*pix.w;
      }
    }
    sum.x/=coeff; sum.y/=coeff; sum.z/=coeff; sum.w/=coeff;

    uchar4 out = {
      static_cast<unsigned char>(sum.x),
      static_cast<unsigned char>(sum.y),
      static_cast<unsigned char>(sum.z),
      static_cast<unsigned char>(sum.w)
    };
    targetData[coord] = out;
}


int main() {

    int width, height, channels;
    uchar4* host_data;

    host_data = reinterpret_cast<uchar4*>(
        stbi_load("tester.bmp", &width, &height, &channels, 4));

    const auto pitchWithPadding = width + 2 * padding;
    const auto sizeWithPadding = pitchWithPadding * (height + 2 * padding);
    const auto pitchWithPaddingInBytes = pitchWithPadding * sizeof(uchar4);
    const auto heightWithPadding = height + 2 * padding;
    const auto widthInBytes = width * sizeof(uchar4);

    // Allocate device memory
    uchar4* paddedData = nullptr;
    uchar4* targetData = nullptr;
    
    size_t devicePitchInBytes;
    size_t targetPitchInBytes;

    hipMallocPitch(&paddedData, &devicePitchInBytes, pitchWithPaddingInBytes, heightWithPadding);
    hipMallocPitch(&targetData, &targetPitchInBytes, pitchWithPaddingInBytes, heightWithPadding);

    const auto skipTopRows = 2 * (devicePitchInBytes / sizeof(uchar4)) + 2;
    
    hipMemset2D(paddedData, devicePitchInBytes, 0, pitchWithPaddingInBytes, heightWithPadding);
    hipMemcpy2D(paddedData + skipTopRows, devicePitchInBytes, host_data, widthInBytes, widthInBytes, height, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, sizeof(kernel), 0, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid(
        (width  + block.x - 1) / block.x,
        (height + block.y - 1) / block.y);

    gaussianBlur<<<grid, block>>>(
        paddedData,
        targetData,
        width,
        height,
        devicePitchInBytes / sizeof(uchar4));
    
    hipMemcpy2D(host_data, widthInBytes, targetData + skipTopRows, targetPitchInBytes, widthInBytes, height, hipMemcpyDeviceToHost);

    stbi_write_bmp("output.bmp", width, height, 4, host_data);

    return 0;
}